#include "hip/hip_runtime.h"
#include "Kernel.cuh"

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void processPixels (hipSurfaceObject_t inputTexture) {
	int x = blockIdx . x * blockDim . x + threadIdx . x ;
	int y = blockIdx . y * blockDim . y + threadIdx . y ;

	uchar4 pixel = tex2D<uchar4>(inputTexture, x, y);
	pixel.x = pixel.x ^ 0xFFu;
	pixel.y = pixel.y ^ 0xFFu;
	pixel.z = pixel.z ^ 0xFFu;
	surf2Dwrite(pixel, inputTexture, x * sizeof(uchar4), y);
}
extern "C" {
	void negatif_parallel(unsigned int textureId, int width, int height) {
		hipGraphicsResource *inputGraphicsResource;

		hipTextureDesc inputTextureDesc;
		memset(&inputTextureDesc, 0, sizeof(inputTextureDesc));
		inputTextureDesc.addressMode[0] = hipAddressModeClamp;
		inputTextureDesc.addressMode[1] = hipAddressModeClamp;

		hipGraphicsGLRegisterImage(&inputGraphicsResource, textureId, GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly);

		hipArray_t inputCudaArray;
		hipGraphicsMapResources(1, &inputGraphicsResource);
		hipGraphicsSubResourceGetMappedArray(&inputCudaArray, inputGraphicsResource, 0, 0);
		hipResourceDesc inputResourceDesc{hipResourceTypeArray, inputCudaArray};
		hipTextureObject_t inputTexture;
		hipCreateTextureObject(&inputTexture, &inputResourceDesc, &inputTextureDesc, NULL);

		const dim3 BLOCK_DIM(32, 32, 1);
		dim3 grid(width / BLOCK_DIM.x, height / BLOCK_DIM.y, 1);
		processPixels << < grid, BLOCK_DIM >> > (inputTexture);
		hipDeviceSynchronize();

		hipDestroyTextureObject(inputTexture);
		hipGraphicsUnmapResources(1, &inputGraphicsResource);
	}
}
